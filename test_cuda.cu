#include "hip/hip_runtime.h"
#include "main.h"

static const char *_cudaGetErrorEnum(hipError_t error) {
    return hipGetErrorName(error);
  }

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
#define cpu_atomic_add32(a, x) __sync_add_and_fetch(a, x)

__global__ void simpleKernel(char *ptr, int sz, char val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (; idx < sz; idx += (gridDim.x * blockDim.x)) {
        ptr[idx] = val;
    }
}

static void barrierWait(volatile int * barrier, volatile int *sense, unsigned int n) {
    int count;

    // check-in
    count = cpu_atomic_add32(barrier, 1);
    if (count == n) *sense = 1;
    while (!*sense);

    count = cpu_atomic_add32(barrier, -1);
    if (count == 0) *sense = 0;
    while(*sense);
}

extern "C" void* test_init(int rank, hipIpcMemHandle_t& handle, hipIpcEventHandle_t& ev_handle, hipEvent_t* event) {
    void* ptr = NULL;
    checkCudaErrors(hipSetDevice(rank));
    if (rank == 0) {
        checkCudaErrors(hipDeviceEnablePeerAccess(1, 0));
    } else {
        checkCudaErrors(hipDeviceEnablePeerAccess(0, 0));
    }
    checkCudaErrors(hipMalloc(&ptr, DATA_SIZE));
    checkCudaErrors(hipIpcGetMemHandle(&handle, ptr));
    checkCudaErrors(hipEventCreate(event, hipEventDisableTiming | hipEventInterprocess));
    checkCudaErrors(hipIpcGetEventHandle(&ev_handle, *event));
    return ptr;
}

extern "C" void run_test(sharedMemoryInfo& info, volatile shmStruct* shm, int size, hipIpcMemHandle_t& handle, hipIpcEventHandle_t& ev_handle, hipIpcMemHandle_t& get_handle, hipIpcEventHandle_t& get_ev_handle, int rank, hipEvent_t ev, void* ptr_t) {
    std::vector<char> verification_buffer(DATA_SIZE);
    hipStream_t stream;
    int blocks = 0;
    int threads = 128;
    hipDeviceProp_t prop;
    if (rank == 0) {
        checkCudaErrors(hipGetDeviceProperties(&prop, 0));
    } else {
        checkCudaErrors(hipGetDeviceProperties(&prop, 1));
    }
    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blocks, simpleKernel, threads, 0));
    blocks *= prop.multiProcessorCount;

    std::vector<void *> ptrs;
    std::vector<hipEvent_t> events;
    for (int i = 0; i < size; i++) {
        void* ptr = NULL;
        hipEvent_t event;
        if (i == rank) {
            ptrs.push_back(ptr_t);
            events.push_back(ev);
            continue;
        } else {
            checkCudaErrors(hipIpcOpenMemHandle(&ptr, get_handle, hipIpcMemLazyEnablePeerAccess));
            checkCudaErrors(hipIpcOpenEventHandle(&event, get_ev_handle));
        }

        ptrs.push_back(ptr);
        events.push_back(event);
    }

    for (int i = 0; i < size; i++) {
        size_t bufferId = (i + rank) % size;
        checkCudaErrors(hipStreamWaitEvent(stream, events[bufferId], 0));

        simpleKernel<<<blocks, threads, 0, stream>>>((char*)ptrs[bufferId], DATA_SIZE, rank);
        checkCudaErrors(hipGetLastError());

        checkCudaErrors(hipEventRecord(events[bufferId], stream));

        barrierWait(&shm->barrier, &shm->sense, (unsigned int)size);
        if (rank == 0) {
            printf("Step %lld done\n", (unsigned long long)i);
        }
    }

    checkCudaErrors(hipStreamWaitEvent(stream, events[rank], 0));
    checkCudaErrors(hipMemcpyAsync(&verification_buffer[0], ptrs[rank], DATA_SIZE, hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    printf("Process %d: verifying...\n", rank);

    char compareId = (char)((rank + 1) % size);
    for (unsigned long long j = 0; j < DATA_SIZE;j++) {
        if (verification_buffer[j] != compareId) {
            printf("Process %d: Verifying mismatch at %lld : %d != %d\n", rank, j, (int)verification_buffer[j], (int)compareId);
        }
    }

    // for (int i = 0; i < size; i++) {
    //     checkCudaErrors(hipIpcCloseMemHandle(ptrs[i]));
    //     checkCudaErrors(hipEventDestroy(events[i]));
    // }

    checkCudaErrors(hipStreamDestroy(stream));

    printf("Process %d complete\n", rank);

}
